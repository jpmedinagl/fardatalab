#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <nvcomp/lz4.h>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << #call << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

void read_file_data(const char* filename, char*& data, size_t& size)
{
    std::ifstream file(filename, std::ios::binary | std::ios::ate);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    size = file.tellg();
    file.seekg(0, std::ios::beg);
    data = new char[size];
    file.read(data, size);
    file.close();
}

void write_file_data(const char* filename, char* data, size_t size)
{
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Failed to open file for writing: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    file.write(data, size);
    file.close();
}

void compression(char* input_data, const size_t in_bytes, char*& compressed_data, size_t& compressed_size)
{
    // Create a CUDA stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Define chunk size and batch size
    const size_t chunk_size = 65536;
    const size_t batch_size = (in_bytes + chunk_size - 1) / chunk_size;

    // Allocate device memory
    char* device_input_data;
    CUDA_CHECK(hipMalloc(&device_input_data, in_bytes));
    CUDA_CHECK(hipMemcpyAsync(device_input_data, input_data, in_bytes, hipMemcpyHostToDevice, stream));

    // Set up uncompressed data pointers and sizes
    size_t* host_uncompressed_bytes;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_bytes, sizeof(size_t) * batch_size));
    for (size_t i = 0; i < batch_size; ++i) {
        host_uncompressed_bytes[i] = (i + 1 < batch_size) ? chunk_size : in_bytes - (chunk_size * i);
    }

    void** host_uncompressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_ptrs, sizeof(void*) * batch_size));
    for (size_t i = 0; i < batch_size; ++i) {
        host_uncompressed_ptrs[i] = device_input_data + chunk_size * i;
    }

    size_t* device_uncompressed_bytes;
    void** device_uncompressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_uncompressed_bytes, sizeof(size_t) * batch_size));
    CUDA_CHECK(hipMalloc(&device_uncompressed_ptrs, sizeof(void*) * batch_size));
    CUDA_CHECK(hipMemcpyAsync(device_uncompressed_bytes, host_uncompressed_bytes, sizeof(size_t) * batch_size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(device_uncompressed_ptrs, host_uncompressed_ptrs, sizeof(void*) * batch_size, hipMemcpyHostToDevice, stream));

    // Allocate space for temporary memory and compressed data
    size_t temp_bytes;
    nvcompBatchedLZ4CompressGetTempSize(batch_size, chunk_size, nvcompBatchedLZ4DefaultOpts, &temp_bytes);
    void* device_temp_ptr;
    CUDA_CHECK(hipMalloc(&device_temp_ptr, temp_bytes));

    size_t max_out_bytes;
    nvcompBatchedLZ4CompressGetMaxOutputChunkSize(chunk_size, nvcompBatchedLZ4DefaultOpts, &max_out_bytes);

    void** host_compressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_compressed_ptrs, sizeof(void*) * batch_size));
    for (size_t i = 0; i < batch_size; ++i) {
        CUDA_CHECK(hipMalloc(&host_compressed_ptrs[i], max_out_bytes));
    }

    void** device_compressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_compressed_ptrs, sizeof(void*) * batch_size));
    CUDA_CHECK(hipMemcpyAsync(device_compressed_ptrs, host_compressed_ptrs, sizeof(void*) * batch_size, hipMemcpyHostToDevice, stream));

    size_t* device_compressed_bytes;
    CUDA_CHECK(hipMalloc(&device_compressed_bytes, sizeof(size_t) * batch_size));

    // Start the timer for GPU compression
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, 0));

    // Perform GPU compression using nvcomp
    nvcompStatus_t comp_res = nvcompBatchedLZ4CompressAsync(
        device_uncompressed_ptrs,
        device_uncompressed_bytes,
        chunk_size,
        batch_size,
        device_temp_ptr,
        temp_bytes,
        device_compressed_ptrs,
        device_compressed_bytes,
        nvcompBatchedLZ4DefaultOpts,
        stream
    );

    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&gpuTime, start, stop));

    if (comp_res != nvcompSuccess) {
        std::cerr << "GPU compression failed!" << std::endl;
        return;
    }

    std::cout << "GPU compression time: " << gpuTime << " ms\n";
    float gpuThroughput = (in_bytes / (1024.0f * 1024.0f)) / (gpuTime / 1000.0f);
    std::cout << "GPU Throughput: " << gpuThroughput << " MB/s\n";

    CUDA_CHECK(hipMemcpyAsync(&compressed_size, device_compressed_bytes, sizeof(size_t), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    
    compressed_data = new char[compressed_size];
    CUDA_CHECK(hipMemcpy(compressed_data, host_compressed_ptrs, compressed_size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // Clean up
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

void decompression(char* compressed_data, size_t compressed_size)
{
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    char* device_compressed_data;
    CUDA_CHECK(hipMalloc(&device_compressed_data, compressed_size));
    CUDA_CHECK(hipMemcpyAsync(device_compressed_data, compressed_data, compressed_size, hipMemcpyHostToDevice, stream));

    const size_t chunk_size = 65536;
    size_t num_chunks = (compressed_size + chunk_size - 1) / chunk_size;

    void** host_compressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_compressed_ptrs, sizeof(void*) * num_chunks));
    for (size_t i = 0; i < num_chunks; ++i) {
        host_compressed_ptrs[i] = device_compressed_data + chunk_size * i;
    }
    
    void** device_compressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_compressed_ptrs, sizeof(void*) * num_chunks));
    CUDA_CHECK(hipMemcpyAsync(device_compressed_ptrs, host_compressed_ptrs, sizeof(void*) * num_chunks, hipMemcpyHostToDevice, stream));

    size_t* device_compressed_bytes;
    CUDA_CHECK(hipMalloc(&device_compressed_bytes, sizeof(size_t) * num_chunks));
    CUDA_CHECK(hipMemcpyAsync(device_compressed_bytes, &compressed_size, sizeof(size_t) * num_chunks, hipMemcpyHostToDevice, stream));

    size_t* device_uncompressed_bytes;
    CUDA_CHECK(hipMalloc(&device_uncompressed_bytes, sizeof(size_t) * num_chunks));

    size_t temp_bytes;
    nvcompBatchedLZ4DecompressGetTempSize(num_chunks, chunk_size, &temp_bytes);
    void* device_temp_ptr;
    CUDA_CHECK(hipMalloc(&device_temp_ptr, temp_bytes));

    size_t uncompressed_size = num_chunks * chunk_size;
    char* device_uncompressed_data;
    CUDA_CHECK(hipMalloc(&device_uncompressed_data, uncompressed_size));

    void** host_uncompressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_ptrs, sizeof(void*) * num_chunks));
    for (size_t i = 0; i < num_chunks; ++i) {
        host_uncompressed_ptrs[i] = device_uncompressed_data + chunk_size * i;
    }
    void** device_uncompressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_uncompressed_ptrs, sizeof(void*) * num_chunks));
    CUDA_CHECK(hipMemcpyAsync(device_uncompressed_ptrs, host_uncompressed_ptrs, sizeof(void*) * num_chunks, hipMemcpyHostToDevice, stream));

    nvcompStatus_t* device_statuses;
    CUDA_CHECK(hipMalloc(&device_statuses, sizeof(nvcompStatus_t) * num_chunks));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, stream));

    nvcompStatus_t decomp_res = nvcompBatchedLZ4DecompressAsync(
        (const void *const *)device_compressed_ptrs,
        device_compressed_bytes,
        device_uncompressed_bytes,
        device_uncompressed_bytes,
        num_chunks,
        device_temp_ptr,
        temp_bytes,
        (void *const *)device_uncompressed_ptrs,
        device_statuses,
        stream
    );
    
    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    float gpuTime = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&gpuTime, start, stop));

    if (decomp_res != nvcompSuccess) {
        std::cerr << "GPU decompression failed!" << std::endl;
        return;
    }

    std::cout << "GPU decompression time: " << gpuTime << " ms\n";
    float compressionRatio = (float)compressed_size / uncompressed_size;
    std::cout << "Compression Ratio: " << compressionRatio << "\n";

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

int main(int argc, char **argv)
{
    if (argc != 2) {
        std::cout << "usage: ./gpu_decomp <file>\n";
        return 1;
    }

    const char* filename = argv[1];
    char* uncompressed_data = nullptr;
    size_t uncompressed_size = 0;

    read_file_data(filename, uncompressed_data, uncompressed_size);

    std::cout << "Starting GPU compression..." << std::endl;

    char* compressed_data = nullptr;
    size_t compressed_size = 0;
    compression(uncompressed_data, uncompressed_size, compressed_data, compressed_size);
    // write_file_data("tmp.bin", compressed_data, compressed_size);

    std::cout << "Starting GPU decompression..." << std::endl;
    decompression(compressed_data, compressed_size);

    delete[] compressed_data;
    return 0;
}
