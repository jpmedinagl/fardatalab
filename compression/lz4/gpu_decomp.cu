#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <nvcomp/lz4.h>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << #call << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

void read_file_data(const char* filename, char*& data, size_t& size)
{
    std::ifstream file(filename, std::ios::binary | std::ios::ate);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    size = file.tellg();
    file.seekg(0, std::ios::beg);
    data = new char[size];
    file.read(data, size);
    file.close();
}

void decompress_example(char* compressed_data, size_t compressed_size)
{
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    char* device_compressed_data;
    CUDA_CHECK(hipMalloc(&device_compressed_data, compressed_size));
    CUDA_CHECK(hipMemcpyAsync(device_compressed_data, compressed_data, compressed_size, hipMemcpyHostToDevice, stream));

    const size_t chunk_size = 65536;
    size_t num_chunks = (compressed_size + chunk_size - 1) / chunk_size;

    void** host_compressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_compressed_ptrs, sizeof(void*) * num_chunks));
    for (size_t i = 0; i < num_chunks; ++i) {
        host_compressed_ptrs[i] = device_compressed_data + chunk_size * i;
    }
    
    void** device_compressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_compressed_ptrs, sizeof(void*) * num_chunks));
    CUDA_CHECK(hipMemcpyAsync(device_compressed_ptrs, host_compressed_ptrs, sizeof(void*) * num_chunks, hipMemcpyHostToDevice, stream));

    size_t* device_compressed_bytes;
    CUDA_CHECK(hipMalloc(&device_compressed_bytes, sizeof(size_t) * num_chunks));
    CUDA_CHECK(hipMemcpyAsync(device_compressed_bytes, &compressed_size, sizeof(size_t) * num_chunks, hipMemcpyHostToDevice, stream));

    size_t* device_uncompressed_bytes;
    CUDA_CHECK(hipMalloc(&device_uncompressed_bytes, sizeof(size_t) * num_chunks));

    size_t temp_bytes;
    nvcompBatchedLZ4DecompressGetTempSize(num_chunks, chunk_size, &temp_bytes);
    void* device_temp_ptr;
    CUDA_CHECK(hipMalloc(&device_temp_ptr, temp_bytes));

    size_t uncompressed_size = num_chunks * chunk_size;
    char* device_uncompressed_data;
    CUDA_CHECK(hipMalloc(&device_uncompressed_data, uncompressed_size));

    void** host_uncompressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_ptrs, sizeof(void*) * num_chunks));
    for (size_t i = 0; i < num_chunks; ++i) {
        host_uncompressed_ptrs[i] = device_uncompressed_data + chunk_size * i;
    }
    void** device_uncompressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_uncompressed_ptrs, sizeof(void*) * num_chunks));
    CUDA_CHECK(hipMemcpyAsync(device_uncompressed_ptrs, host_uncompressed_ptrs, sizeof(void*) * num_chunks, hipMemcpyHostToDevice, stream));

    nvcompStatus_t* device_statuses;
    CUDA_CHECK(hipMalloc(&device_statuses, sizeof(nvcompStatus_t) * num_chunks));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, stream));

    nvcompStatus_t decomp_res = nvcompBatchedLZ4DecompressAsync(
        (const void *const *)device_compressed_ptrs,
        device_compressed_bytes,
        device_uncompressed_bytes,
        device_uncompressed_bytes,
        num_chunks,
        device_temp_ptr,
        temp_bytes,
        (void *const *)device_uncompressed_ptrs,
        device_statuses,
        stream
    );
    
    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    float gpuTime = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&gpuTime, start, stop));

    if (decomp_res != nvcompSuccess) {
        std::cerr << "GPU decompression failed!" << std::endl;
        return;
    }

    std::cout << "GPU decompression time: " << gpuTime << " ms\n";
    float compressionRatio = (float)compressed_size / uncompressed_size;
    std::cout << "Compression Ratio: " << compressionRatio << "\n";

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

int main(int argc, char **argv)
{
    if (argc != 2) {
        std::cout << "usage: ./gpu_decomp <compressed_file>\n";
        return 1;
    }

    const char* filename = argv[1];
    char* compressed_data = nullptr;
    size_t compressed_size = 0;

    read_file_data(filename, compressed_data, compressed_size);

    std::cout << "Starting GPU decompression..." << std::endl;
    decompress_example(compressed_data, compressed_size);

    delete[] compressed_data;
    return 0;
}
