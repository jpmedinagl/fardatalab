#include <iostream>
#include <fstream>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <nvcomp/lz4.h>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << #call << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

void read_file_data(const char* filename, char*& data, size_t& size)
{
    // Open the file in binary mode
    std::ifstream file(filename, std::ios::binary | std::ios::ate);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    // Get the file size and allocate buffer for data
    size = file.tellg();
    file.seekg(0, std::ios::beg);
    data = new char[size];

    // Read the file content into the buffer
    file.read(data, size);
    file.close();
}

void write_file_data(const char* filename, char* data, size_t size)
{
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Failed to open file for writing: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    file.write(data, size);
    file.close();
}

void compression(char* input_data, const size_t in_bytes)
{
    // Create a CUDA stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Define chunk size and batch size
    const size_t chunk_size = 65536;
    const size_t batch_size = (in_bytes + chunk_size - 1) / chunk_size;

    // Allocate device memory
    char* device_input_data;
    CUDA_CHECK(hipMalloc(&device_input_data, in_bytes));
    CUDA_CHECK(hipMemcpyAsync(device_input_data, input_data, in_bytes, hipMemcpyHostToDevice, stream));

    // Set up uncompressed data pointers and sizes
    size_t* host_uncompressed_bytes;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_bytes, sizeof(size_t) * batch_size));
    for (size_t i = 0; i < batch_size; ++i) {
        host_uncompressed_bytes[i] = (i + 1 < batch_size) ? chunk_size : in_bytes - (chunk_size * i);
    }

    void** host_uncompressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_ptrs, sizeof(void*) * batch_size));
    for (size_t i = 0; i < batch_size; ++i) {
        host_uncompressed_ptrs[i] = device_input_data + chunk_size * i;
    }

    size_t* device_uncompressed_bytes;
    void** device_uncompressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_uncompressed_bytes, sizeof(size_t) * batch_size));
    CUDA_CHECK(hipMalloc(&device_uncompressed_ptrs, sizeof(void*) * batch_size));
    CUDA_CHECK(hipMemcpyAsync(device_uncompressed_bytes, host_uncompressed_bytes, sizeof(size_t) * batch_size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(device_uncompressed_ptrs, host_uncompressed_ptrs, sizeof(void*) * batch_size, hipMemcpyHostToDevice, stream));

    // Allocate space for temporary memory and compressed data
    size_t temp_bytes;
    nvcompBatchedLZ4CompressGetTempSize(batch_size, chunk_size, nvcompBatchedLZ4DefaultOpts, &temp_bytes);
    void* device_temp_ptr;
    CUDA_CHECK(hipMalloc(&device_temp_ptr, temp_bytes));

    size_t max_out_bytes;
    nvcompBatchedLZ4CompressGetMaxOutputChunkSize(chunk_size, nvcompBatchedLZ4DefaultOpts, &max_out_bytes);

    void** host_compressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_compressed_ptrs, sizeof(void*) * batch_size));
    for (size_t i = 0; i < batch_size; ++i) {
        CUDA_CHECK(hipMalloc(&host_compressed_ptrs[i], max_out_bytes));
    }

    void** device_compressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_compressed_ptrs, sizeof(void*) * batch_size));
    CUDA_CHECK(hipMemcpyAsync(device_compressed_ptrs, host_compressed_ptrs, sizeof(void*) * batch_size, hipMemcpyHostToDevice, stream));

    size_t* device_compressed_bytes;
    CUDA_CHECK(hipMalloc(&device_compressed_bytes, sizeof(size_t) * batch_size));

    // Start the timer for GPU compression
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, 0));

    // Perform GPU compression using nvcomp
    nvcompStatus_t comp_res = nvcompBatchedLZ4CompressAsync(
        device_uncompressed_ptrs,
        device_uncompressed_bytes,
        chunk_size,
        batch_size,
        device_temp_ptr,
        temp_bytes,
        device_compressed_ptrs,
        device_compressed_bytes,
        nvcompBatchedLZ4DefaultOpts,
        stream
    );

    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&gpuTime, start, stop));

    if (comp_res != nvcompSuccess) {
        std::cerr << "GPU compression failed!" << std::endl;
        return;
    }

    std::cout << "GPU compression time: " << gpuTime << " ms\n";
    float gpuThroughput = (in_bytes / (1024.0f * 1024.0f)) / (gpuTime / 1000.0f);
    std::cout << "GPU Throughput: " << gpuThroughput << " MB/s\n";

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // Retrieve the compressed data from device to host
    char* compressed_data = new char[batch_size * max_out_bytes];
    for (size_t i = 0; i < batch_size; ++i) {
        CUDA_CHECK(hipMemcpy(compressed_data + i * max_out_bytes, host_compressed_ptrs[i], max_out_bytes, hipMemcpyDeviceToHost));
    }

    // Write compressed data to a file
    write_file_data("temp", compressed_data, batch_size * max_out_bytes);

    // Clean up
    delete[] compressed_data;
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

int main(int argc, char ** argv)
{
    if (argc != 2) {
        std::cout << "usage: ./gpu_comp <file>\n";
        return 1;
    }

    const char* filename = argv[1];  // Change this to your data file
    char* uncompressed_data = nullptr;
    size_t data_size = 0;

    // Read the data from the file
    read_file_data(filename, uncompressed_data, data_size);

    // Run GPU compression
    std::cout << "Starting GPU compression..." << std::endl;
    compression(uncompressed_data, data_size);

    // Clean up
    delete[] uncompressed_data;

    return 0;
}
