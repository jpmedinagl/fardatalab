#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <nvcomp/lz4.h>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << #call << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

void read_file_data(const char* filename, char*& data, size_t& size)
{
    // Open the file in binary mode
    std::ifstream file(filename, std::ios::binary | std::ios::ate);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    // Get the file size and allocate buffer for data
    size = file.tellg();
    file.seekg(0, std::ios::beg);
    data = new char[size];

    // Read the file content into the buffer
    file.read(data, size);
    file.close();
}

void write_file_data(const char* filename, char* data, size_t size)
{
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Failed to open file for writing: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    file.write(data, size);
    file.close();
}

void compress_chunk(char* input_data_chunk, const size_t chunk_size, hipStream_t stream) {
    // Allocate device memory
    CUDA_CHECK(hipStreamCreate(&stream));

    char* device_input_data;
    CUDA_CHECK(hipMalloc(&device_input_data, chunk_size));
    CUDA_CHECK(hipMemcpy(device_input_data, input_data_chunk, chunk_size, hipMemcpyHostToDevice));

    // Set up uncompressed data pointers and sizes
    size_t* host_uncompressed_bytes;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_bytes, sizeof(size_t)));
    host_uncompressed_bytes[0] = chunk_size;

    void** host_uncompressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_ptrs, sizeof(void*)));
    host_uncompressed_ptrs[0] = device_input_data;

    size_t* device_uncompressed_bytes;
    void** device_uncompressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_uncompressed_bytes, sizeof(size_t)));
    CUDA_CHECK(hipMalloc(&device_uncompressed_ptrs, sizeof(void*)));
    CUDA_CHECK(hipMemcpy(device_uncompressed_bytes, host_uncompressed_bytes, sizeof(size_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(device_uncompressed_ptrs, host_uncompressed_ptrs, sizeof(void*), hipMemcpyHostToDevice));

    // Allocate space for temporary memory and compressed data
    size_t temp_bytes;
    nvcompBatchedLZ4CompressGetTempSize(1, chunk_size, nvcompBatchedLZ4DefaultOpts, &temp_bytes);
    void* device_temp_ptr;
    CUDA_CHECK(hipMalloc(&device_temp_ptr, temp_bytes));

    size_t max_out_bytes;
    nvcompBatchedLZ4CompressGetMaxOutputChunkSize(chunk_size, nvcompBatchedLZ4DefaultOpts, &max_out_bytes);

    void* device_compressed_ptr;
    CUDA_CHECK(hipMalloc(&device_compressed_ptr, max_out_bytes));

    size_t* device_compressed_bytes;
    CUDA_CHECK(hipMalloc(&device_compressed_bytes, sizeof(size_t)));

    // Start the timer for GPU compression
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, 0));

    // Perform GPU compression using nvcomp
    nvcompStatus_t comp_res = nvcompBatchedLZ4CompressAsync(
        device_uncompressed_ptrs,
        device_uncompressed_bytes,
        chunk_size,
        1, // Only one batch in each chunk call
        device_temp_ptr,
        temp_bytes,
        &device_compressed_ptr,
        device_compressed_bytes,
        nvcompBatchedLZ4DefaultOpts,
        stream
    );

    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&gpuTime, start, stop));

    if (comp_res != nvcompSuccess) {
        std::cerr << "GPU compression failed!" << std::endl;
        return;
    }

    std::cout << "GPU compression time for chunk: " << gpuTime << " ms\n";
    float gpuThroughput = (chunk_size / (1024.0f * 1024.0f)) / (gpuTime / 1000.0f);
    std::cout << "GPU Throughput: " << gpuThroughput << " MB/s\n";

    // Retrieve the compressed data from device to host
    // char* compressed_data = new char[max_out_bytes];
    // CUDA_CHECK(hipMemcpy(compressed_data, device_compressed_ptr, max_out_bytes, hipMemcpyDeviceToHost));

    // Optionally, you can write the compressed data for each chunk to a file or further process
    // write_file_data("temp_chunk", compressed_data, max_out_bytes);

    // Clean up
    // delete[] compressed_data;
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(device_input_data));
    CUDA_CHECK(hipFree(device_temp_ptr));
    CUDA_CHECK(hipFree(device_compressed_ptr));
    CUDA_CHECK(hipFree(device_compressed_bytes));
    CUDA_CHECK(hipFree(device_uncompressed_bytes));
    CUDA_CHECK(hipFree(device_uncompressed_ptrs));

    CUDA_CHECK(hipStreamSynchronize(stream));
}

void compression(char* input_data, const size_t in_bytes) {
    // Create a CUDA stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Define chunk size
    const size_t chunk_size = 65536;
    
    // Calculate the number of sections needed based on chunk size
    const size_t num_chunks = (in_bytes + chunk_size - 1) / chunk_size;

    // Iterate over the input data in chunks
    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        size_t chunk_offset = chunk_idx * chunk_size;
        size_t chunk_data_size = std::min(chunk_size, in_bytes - chunk_offset);

        // Call compression for the current chunk of data
        compress_chunk(input_data + chunk_offset, chunk_data_size, stream);
    }

    // Cleanup stream
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

int main(int argc, char ** argv)
{
    if (argc != 2) {
        std::cout << "usage: ./gpu_comp <file>\n";
        return 1;
    }

    const char* filename = argv[1];  // Change this to your data file
    char* uncompressed_data = nullptr;
    size_t data_size = 0;

    // Read the data from the file
    read_file_data(filename, uncompressed_data, data_size);

    // Run GPU compression
    std::cout << "Starting GPU compression..." << std::endl;
    compression(uncompressed_data, data_size);

    // Clean up
    delete[] uncompressed_data;

    return 0;
}
