#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <nvcomp/lz4.h>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << #call << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

void read_file_data(const char* filename, char*& data, size_t& size)
{
    // Open the file in binary mode
    std::ifstream file(filename, std::ios::binary | std::ios::ate);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    // Get the file size and allocate buffer for data
    size = file.tellg();
    file.seekg(0, std::ios::beg);
    data = new char[size];

    // Read the file content into the buffer
    file.read(data, size);
    file.close();
}

void write_file_data(const char* filename, char* data, size_t size)
{
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Failed to open file for writing: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    file.write(data, size);
    file.close();
}

int compress_chunk(char* input_data, const size_t chunk_size, hipStream_t stream)
{
    // Create a CUDA stream
    // hipStream_t stream;
    // CUDA_CHECK(hipStreamCreate(&stream));

    // Define chunk size and batch size
    // const size_t chunk_size = 65536;
    // const size_t batch_size = (in_bytes + chunk_size - 1) / chunk_size;

    const size_t batch_size = 1;
    const size_t in_bytes = chunk_size;

    // Allocate device memory
    char* device_input_data;
    CUDA_CHECK(hipMalloc(&device_input_data, in_bytes));
    CUDA_CHECK(hipMemcpy(device_input_data, input_data, in_bytes, hipMemcpyHostToDevice));

    // Set up uncompressed data pointers and sizes
    size_t* host_uncompressed_bytes;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_bytes, sizeof(size_t) * batch_size));
    for (size_t i = 0; i < batch_size; ++i) {
        host_uncompressed_bytes[i] = (i + 1 < batch_size) ? chunk_size : in_bytes - (chunk_size * i);
    }

    void** host_uncompressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_uncompressed_ptrs, sizeof(void*) * batch_size));
    for (size_t i = 0; i < batch_size; ++i) {
        host_uncompressed_ptrs[i] = device_input_data + chunk_size * i;
    }

    size_t* device_uncompressed_bytes;
    void** device_uncompressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_uncompressed_bytes, sizeof(size_t) * batch_size));
    CUDA_CHECK(hipMalloc(&device_uncompressed_ptrs, sizeof(void*) * batch_size));
    CUDA_CHECK(hipMemcpy(device_uncompressed_bytes, host_uncompressed_bytes, sizeof(size_t) * batch_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(device_uncompressed_ptrs, host_uncompressed_ptrs, sizeof(void*) * batch_size, hipMemcpyHostToDevice));

    // Allocate space for temporary memory and compressed data
    size_t temp_bytes;
    nvcompBatchedLZ4CompressGetTempSize(batch_size, chunk_size, nvcompBatchedLZ4DefaultOpts, &temp_bytes);
    void* device_temp_ptr;
    CUDA_CHECK(hipMalloc(&device_temp_ptr, temp_bytes));

    size_t max_out_bytes;
    nvcompBatchedLZ4CompressGetMaxOutputChunkSize(chunk_size, nvcompBatchedLZ4DefaultOpts, &max_out_bytes);

    void** host_compressed_ptrs;
    CUDA_CHECK(hipHostMalloc(&host_compressed_ptrs, sizeof(void*) * batch_size));
    for (size_t i = 0; i < batch_size; ++i) {
        CUDA_CHECK(hipMalloc(&host_compressed_ptrs[i], max_out_bytes));
    }

    void** device_compressed_ptrs;
    CUDA_CHECK(hipMalloc(&device_compressed_ptrs, sizeof(void*) * batch_size));
    CUDA_CHECK(hipMemcpy(device_compressed_ptrs, host_compressed_ptrs, sizeof(void*) * batch_size, hipMemcpyHostToDevice));

    size_t* device_compressed_bytes;
    CUDA_CHECK(hipMalloc(&device_compressed_bytes, sizeof(size_t) * batch_size));

    // Start the timer for GPU compression
    // hipEvent_t start, stop;
    // float gpuTime = 0.0f;
    // CUDA_CHECK(hipEventCreate(&start));
    // CUDA_CHECK(hipEventCreate(&stop));
    // CUDA_CHECK(hipEventRecord(start, 0));

    // Perform GPU compression using nvcomp
    nvcompStatus_t comp_res = nvcompBatchedLZ4CompressAsync(
        device_uncompressed_ptrs,
        device_uncompressed_bytes,
        chunk_size,
        batch_size,
        device_temp_ptr,
        temp_bytes,
        device_compressed_ptrs,
        device_compressed_bytes,
        nvcompBatchedLZ4DefaultOpts,
        stream
    );

    // CUDA_CHECK(hipEventRecord(stop, 0));
    // CUDA_CHECK(hipEventSynchronize(stop));
    // CUDA_CHECK(hipEventElapsedTime(&gpuTime, start, stop));

    if (comp_res != nvcompSuccess) {
        std::cerr << "GPU compression failed!" << std::endl;
        return -1;
    }

    // std::cout << "GPU compression time: " << gpuTime << " ms\n";
    // float gpuThroughput = (in_bytes / (1024.0f * 1024.0f)) / (gpuTime / 1000.0f);
    // std::cout << "GPU Throughput: " << gpuThroughput << " MB/s\n";

    // CUDA_CHECK(hipEventDestroy(start));
    // CUDA_CHECK(hipEventDestroy(stop));

    // Retrieve the compressed data from device to host
    // char* compressed_data = new char[batch_size * max_out_bytes];
    // for (size_t i = 0; i < batch_size; ++i) {
    //     CUDA_CHECK(hipMemcpy(compressed_data + i * max_out_bytes, host_compressed_ptrs[i], max_out_bytes, hipMemcpyDeviceToHost));
    // }

    // Write compressed data to a file
    // write_file_data("temp", compressed_data, batch_size * max_out_bytes);

    // Retrieve the total compressed size from the device
    // size_t total_compressed_size = 0;
    // size_t* host_compressed_bytes = new size_t[batch_size];
    // CUDA_CHECK(hipMemcpy(host_compressed_bytes, device_compressed_bytes, sizeof(size_t) * batch_size, hipMemcpyDeviceToHost));

    // // Sum up the sizes of all compressed chunks
    // for (size_t i = 0; i < batch_size; ++i) {
    //     total_compressed_size += host_compressed_bytes[i];
    // }

    // Calculate the GPU compression ratio
    // float gpu_ratio = (float) in_bytes / total_compressed_size;
    // std::cout << "GPU ratio: " << gpu_ratio << "\n";

    // delete[] host_compressed_bytes;

    // Clean up
    // delete[] compressed_data;
    // CUDA_CHECK(hipStreamSynchronize(stream));
    // CUDA_CHECK(hipStreamDestroy(stream));
    return 0;
}

int compression(char* input_data, const size_t in_bytes) {
    // Create a CUDA stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Define chunk size
    const size_t chunk_size = 65536;
    
    // Calculate the number of sections needed based on chunk size
    const size_t num_chunks = (in_bytes + chunk_size - 1) / chunk_size;

    hipEvent_t start_o, stop_o;
    float gpuTime = 0.0f;
    CUDA_CHECK(hipEventCreate(&start_o));
    CUDA_CHECK(hipEventCreate(&stop_o));
    CUDA_CHECK(hipEventRecord(start_o, 0));

    // Iterate over the input data in chunks
    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        size_t chunk_offset = chunk_idx * chunk_size;
        size_t chunk_data_size = std::min(chunk_size, in_bytes - chunk_offset);

        // Call compression for the current chunk of data
        if (compress_chunk(input_data + chunk_offset, chunk_data_size, stream) == -1) {
            return -1;
        }
    }

    CUDA_CHECK(hipEventRecord(stop_o, 0));
    CUDA_CHECK(hipEventSynchronize(stop_o));
    CUDA_CHECK(hipEventElapsedTime(&gpuTime, start_o, stop_o));

    std::cout << "GPU compression time: " << gpuTime << " ms\n";
    float gpuThroughput = (in_bytes / (1024.0f * 1024.0f)) / (gpuTime / 1000.0f);
    std::cout << "GPU Throughput: " << gpuThroughput << " MB/s\n";

    // Cleanup stream
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));

    return 0;
}

int main(int argc, char ** argv)
{
    if (argc != 2) {
        std::cout << "usage: ./gpu_comp <file>\n";
        return 1;
    }

    const char* filename = argv[1];  // Change this to your data file
    char* uncompressed_data = nullptr;
    size_t data_size = 0;

    // Read the data from the file
    read_file_data(filename, uncompressed_data, data_size);

    // Run GPU compression
    std::cout << "Starting GPU compression..." << std::endl;
    compression(uncompressed_data, data_size);

    // Clean up
    delete[] uncompressed_data;

    return 0;
}
