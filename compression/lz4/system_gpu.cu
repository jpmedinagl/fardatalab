#include <iostream>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << #call << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

int main() {
    size_t free_mem, total_mem;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    std::cout << "Free memory: " << free_mem << " bytes\n";
}