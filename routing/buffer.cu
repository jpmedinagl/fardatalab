#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 1024
#define BATCH 256

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

int main() 
{
    int *d_src, *d_dst;  // Device pointers
    int *h_src, *h_dst;  // Host buffers

    h_src = new int[SIZE];
    h_dst = new int[SIZE];

    // Random data
    for (int i = 0; i < SIZE; i++) {
        h_src[i] = i;
    }

    // Set up GPU 0
    CUDA_CHECK(hipSetDevice(0));
    CUDA_CHECK(hipMalloc((void**)&d_src, SIZE * sizeof(int)));

    // Copy data to GPU 0
    CUDA_CHECK(hipMemcpy(d_src, h_src, SIZE * sizeof(int), hipMemcpyHostToDevice));

    // Set up GPU 1
    CUDA_CHECK(hipSetDevice(1));
    CUDA_CHECK(hipMalloc((void **)&d_dst, SIZE * sizeof(int)));

    
    // Transfer GPU 0 to GPU 1
    CUDA_CHECK(hipMemcpyPeer(d_dst, 1, d_src, 0, SIZE * sizeof(int)));


    // Verify data - copy GPU 1 back to host
    CUDA_CHECK(hipMemcpy(h_dst, d_dst, SIZE * sizeof(int), hipMemcpyDeviceToHost));

    bool success = true;
    for (int i = 0; i < SIZE; i++) {
        if (h_src[i] != h_dst[i]) {
            success = false;
            std::cerr << "Mismatch at index " << i << ": " << h_src[i] << " != " << h_dst[i] << std::endl;
            break;
        }
    }

    std::cout << (success ? "Data transfer successful!" : "Data transfer failed!") << std::endl;

    CUDA_CHECK(hipFree(d_src));
    CUDA_CHECK(hipFree(d_dst));
    delete[] h_src;
    delete[] h_dst;

    return 0;
}