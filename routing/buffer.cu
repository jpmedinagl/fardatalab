#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 1024
#define BATCH 256

void checkCuda(hipError_t err, const char * msg) 
{
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() 
{
    int *d_src, *d_dst;  // Device pointers
    int *h_src, *h_dst;  // Host buffers

    h_src = new int[SIZE];
    h_dst = new int[SIZE];

    // Random data
    for (int i = 0; i < SIZE; i++) {
        h_src[i] = i;
    }

    // Set up GPU 0
    checkCuda(hipSetDevice(0), "Set GPU 0");
    checkCuda(hipMalloc((void**)&d_src, SIZE * sizeof(int)), "Allocate GPU 0 memory");

    // Copy data to GPU 0
    checkCuda(hipMemcpy(d_src, h_src, SIZE * sizeof(int), hipMemcpyHostToDevice), "Memcpy to GPU 0");

    // Set up GPU 1
    checkCuda(hipSetDevice(1), "Set GPU 1");
    checkCuda(hipMalloc((void **)&d_dst, SIZE * sizeof(int)), "Allocate GPU 1 memory");

    
    // Transfer GPU 0 to GPU 1
    checkCuda(hipMemcpyPeer(d_dst, 1, d_src, 0, SIZE * sizeof(int)), "Memcpy Peer 0 to 1");


    // Verify data - copy GPU 1 back to host
    checkCuda(hipMemcpy(h_dst, d_dst, SIZE * sizeof(int), hipMemcpyDeviceToHost), "Memcpy back to host");

    bool success = true;
    for (int i = 0; i < SIZE; i++) {
        if (h_src[i] != h_dst[i]) {
            success = false;
            std::cerr << "Mismatch at index " << i << ": " << h_src[i] << " != " << h_dst[i] << std::endl;
            break;
        }
    }

    std::cout << (success ? "Data transfer successful!" : "Data transfer failed!") << std::endl;

    checkCuda(hipFree(d_src), "Free GPU 0 memory");
    checkCuda(hipFree(d_dst), "Free GPU 1 memory");
    delete[] h_src;
    delete[] h_dst;

    return 0;
}